
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__    // <--- writing a kernel function to be run on the gpu (called on host)
void saveIDs(int *idsOut){
     //int tid = threadIdx.x;
     //    int bidx = blockIdx.x;
     //	     int bdim = blockDim.x;
     //	         int globaltid;
     //
     //globaltid = blockIdx.x * blockDim.x + threadIdx.x;
    
     const int tid = threadIdx.x;  // <-- multi-dimensional; just getting one dim right now

    idsOut[tid] = tid;  // put my gpu thread id into the vector
}

// __host__   default for all functions
// __device__ can be invoked on GPU side (called on device and runs on device - kernel helpers)

__global__    // <--- writing a kernel function to be run on the gpu (called on host)
void steady(double *plate1, double *plate2, double *maxDiff, int &rows, int &cols, double &epsilon, int &iteration){
     double diff = 0;
     int globaltid;
     globaltid = blockIdx.x * blockDim.x + threadIdx.x;
     //if (globaltid == -1){
     //  for(int i=0;i<rows;i++){
     //	 for (int j=0;j<cols;j++){
     //	   printf("%4.4f ", *(plate1 + i*cols + j));
     //	 }
     //	 printf("\n");
     //  }
     //}
     //if (globaltid > 4990)
     //  printf("MADE IT %d\n",globaltid);
     //return;
     //if (globaltid < rows-2){
       //printf("%d: %4.2f %d %d %1.2f %d\n", globaltid, *(plate1 + globaltid*cols), rows, cols, epsilon, iteration);
     if (globaltid < rows-2){
       maxDiff[globaltid]=0;
       //printf("%d: %d %d\n", globaltid, iteration, iteration%2);
       if (iteration%2==0){
	 for(int i=1;i<cols-1;i++){
	   *(plate2 + (globaltid+1)*cols + i) = 0.25 * (*(plate1 + (globaltid+1)*cols + i-1) +
							*(plate1 + (globaltid+1)*cols + i+1) +
							*(plate1 + (globaltid+1-1)*cols + i) +
							*(plate1 + (globaltid+1+1)*cols + i));

	   diff = abs(*(plate2 + (globaltid+1)*cols + i) - *(plate1 + (globaltid+1)*cols + i));
	   if(maxDiff[globaltid]<diff)
	     maxDiff[globaltid]=diff;
	 
	 }
	 //printf("%d: %4.4f %4.4f\n", globaltid, maxDiff[globaltid],diff);
	
       }else{

	 for(int i=1;i<cols-1;i++){
	   //val = *(plate2 (globaltid+1)*cols + i-1) ;
	   *(plate1 + (globaltid+1)*cols + i) = 0.25 * (*(plate2 + (globaltid+1)*cols + i-1) +
							*(plate2 + (globaltid+1)*cols + i+1) +
							*(plate2 + (globaltid+1-1)*cols + i) +
							*(plate2 + (globaltid+1+1)*cols + i));

	   diff = abs(*(plate1 + (globaltid+1)*cols + i) - *(plate2 + (globaltid+1)*cols + i));
	   if(maxDiff[globaltid]<diff)
	     maxDiff[globaltid]=diff;
	 
	 }
       }
     }
     //printf("%d: %4.4f %4.4f\n", globaltid, maxDiff[globaltid],diff);

     //}

     //double **myplate = plate1 + globaltid*cols);
     //printf("%d: %4.2f %d %d %1.2f %d\n", globaltid, *(plate1 + globaltid*cols), rows, cols, epsilon, iteration);
     //printf("%d: %4.2f %d %d %1.2f %d\n", globaltid, 49.6, rows, cols, epsilon, iteration);
     //printf("%d: \n", globaltid);

}

int main(int argc, char *argv[])
{
	
    if (argc < 8){
       printf("Too Few Params\n");
       return -1;
    }
    int rows, *drows;
    int cols, *dcols;
    int power=0, iteration=0,*diteration;
    double top;
    double myleft;
    double myright;
    double bottom;
    double epsilon, *depsilon;

    double **hplate1;
    double **hplate2;
    double *dplate1;
    double *dplate2;
    double *maxDiff, *dmaxDiff, totalmaxDiff;
    int blocks, threadsPerBlock;
    
    rows = atoi(argv[1]);
    cols = atoi(argv[2]);
    top = atof(argv[3]);
    myleft = atof(argv[4]);
    myright = atof(argv[5]);
    bottom = atof(argv[6]);
    epsilon = atof(argv[7]);

    //double numerator = ((cols-2)*top + (rows-1)*(myleft+myright) + bottom*cols);
    //double demon = (cols-2) + 2*(rows-1) + cols;
    double start = ((cols-2)*top + (rows-1)*(myleft+myright) + bottom*cols)  /  ((cols-2) + 2*(rows-1) + cols);
    //printf("%4.2f\n", start);


    hplate1 = (double **)malloc(rows*sizeof(double *));
    hplate1[0] = (double *)malloc(rows*cols*sizeof(double));
    hplate2 = (double **)malloc(rows*sizeof(double *));
    hplate2[0] = (double *)malloc(rows*cols*sizeof(double));
    maxDiff = (double *)malloc((rows-2)*sizeof(double));
    
    //void steady(double *plate1, double *plate2, double *maxDiff, int &rows, int &cols, double &epsilon, int &iteration){
    hipMalloc(&dplate1,rows*cols*sizeof(double));
    hipMalloc(&dplate2,rows*cols*sizeof(double));
    hipMalloc(&dmaxDiff,(rows-2)*sizeof(double));
    hipMalloc(&drows,sizeof(int));
    hipMalloc(&dcols,sizeof(int));
    hipMalloc(&depsilon,sizeof(double));
    hipMalloc(&diteration,sizeof(int));
    
    
    
    
    for(int i=0;i<rows;i++){
      hplate1[i] = (*hplate1 + i*cols);
      hplate2[i] = (*hplate2 + i*cols);

    }
  
    for(int r=0;r<rows-1;r++){
      hplate1[r][0] = myleft;
      hplate1[r][cols-1] = myright;
      hplate2[r][0] = myleft;
      hplate2[r][cols-1] = myright;

    }
    for(int c=0;c<cols;c++){
       if (c==0 || c==cols-1){
       	  hplate1[rows-1][c] = bottom;
	  hplate2[rows-1][c] = bottom;

       }else{
          hplate1[0][c] = top;
	  hplate1[rows-1][c] = bottom;
	  hplate2[0][c] = top;
	  hplate2[rows-1][c] = bottom;
       }
    }

    for(int r=1;r<rows-1;r++){
       for(int c=1;c<cols-1;c++){
	  hplate1[r][c] = start;
	  hplate2[r][c] = start;
       }
    }
			  
    //void steady(double *plate1, double *plate2, double *maxDiff, int &rows, int &cols, double &epsilon, int &iteration){
    hipMemcpy(dplate1, hplate1[0], rows*cols*sizeof(double),  hipMemcpyHostToDevice);
    hipMemcpy(dplate2, hplate2[0], rows*cols*sizeof(double),  hipMemcpyHostToDevice);
    hipMemcpy(drows, &rows, sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(dcols, &cols, sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(depsilon, &epsilon, sizeof(double),  hipMemcpyHostToDevice);
    //steady();
    blocks = (int)((rows-2)/1024) + 1;
    if ((rows-2)%blocks == 0)
      threadsPerBlock = (int)((rows-2)/blocks);
    else
      threadsPerBlock = (int)((rows-2)/blocks) + blocks;

    printf("blocks %d, threads per block %d\n\n", blocks, threadsPerBlock);
	
    do{
      hipMemcpy(diteration, &iteration, sizeof(int),  hipMemcpyHostToDevice);
    
      steady<<< blocks, threadsPerBlock >>>(dplate1, dplate2, dmaxDiff, *drows, *dcols, *depsilon, *diteration);
      hipDeviceSynchronize();
      hipMemcpy(maxDiff, dmaxDiff, (rows-2)*sizeof(double),  hipMemcpyDeviceToHost);
      totalmaxDiff = 0;
      for (int i=0;i<rows-2;i++){
	//printf("maxDiff[%d] = %f\n", i, maxDiff[i]);
	if(maxDiff[i]>totalmaxDiff)
	  totalmaxDiff=maxDiff[i];
	
      }

      if(iteration == (int)pow(2.0,power)){
	power+=1;
	printf("%d: %f\n",iteration,totalmaxDiff);
      }
      //if(iteration == 0){
      //	printf("%d: %f\n",iteration,totalmaxDiff);
      //
      //
      //}//
      iteration+=1;
      //break;
    }while(totalmaxDiff>epsilon);

    printf("%d: %4.4f\n",iteration-1,totalmaxDiff);
    
    
    free(hplate1);
    free(hplate2);
    free(maxDiff);
    hipFree(dplate1);
    hipFree(dplate2);
    hipFree(dmaxDiff);
    hipFree(drows);
    hipFree(dcols);
    hipFree(depsilon);
    hipFree(diteration);
    //cudaFree(dIDs);
    //hplate1 = (double **)malloc(rows*sizeof(double *));
    //hplate1[0] = (double *)malloc(rows*cols*sizeof(double));
    //hplate2 = (double **)malloc(rows*sizeof(double *));
    //hplate2[0] = (double *)malloc(rows*cols*sizeof(double));
    //maxDiff = (double *)malloc((rows-2)*sizeof(double));
    //
    ////void steady(double *plate1, double *plate2, double *maxDiff, int &rows, int &cols, double &epsilon, int &iteration){
    //cudaMalloc(&dplate1,rows*cols*sizeof(double));
    //cudaMalloc(&dplate2,rows*cols*sizeof(double));
    //cudaMalloc(&dmaxDiff,(rows-2)*sizeof(double));
    //cudaMalloc(&drows,sizeof(int));
    //cudaMalloc(&dcols,sizeof(int));
    //cudaMalloc(&depsilon,sizeof(double));
    //cudaMalloc(&diteration,sizeof(int));
    

    printf("done\n");
    hipDeviceReset();
			
    return 0;
}
    
